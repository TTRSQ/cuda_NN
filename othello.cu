#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <queue>
#include <map>
#include <cmath>
#include <random>
#include <time.h>
#include "network.cu"
#include "matplotlib.hpp"

using namespace std;

#define yet 0
#define wht 1
#define blk 2

int direction[8][2] = {
    {0,1}, {0,-1}, {1,0}, {-1,0},
    {1,1}, {1,-1}, {-1,-1}, {-1,1}
};

std::map<int, string> color_string;

int ban[8][8] = {};

void init_ban(){
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            ban[i][j] = 0;
        }
    }
    ban[3][3] = wht;
    ban[4][4] = wht;
    ban[3][4] = blk;
    ban[4][3] = blk;
}

void cpytoarray(int array[8][8]){
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            array[i][j] = ban[i][j];
        }
    }
}

void cpytovector(std::vector<double> &vec, int array[8][8], int pcol){
    vec.resize(128);
    for (int k = 0; k < 2; k++) {
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                vec[64*k+8*i+j] = (array[i][j] == pcol)? 1.0: 0.0;
            }
        }
        pcol = (pcol == wht)? blk: wht;
    }
}

bool update(int x, int y, int col, int dir, int depth, int banmen[8][8]){
    x += direction[dir][0];
    y += direction[dir][1];

    if ((x < 0 || 7 < x) || (y < 0 || 7 < y) ) return false;
    if (banmen[y][x] == yet) return false;

    if (banmen[y][x] == col) {
        if(depth == 0) return false;
        else return true;
    }else{
        if(update(x, y, col, dir, depth+1, banmen)){
            banmen[y][x] = col;
            return true;
        }else{
            return false;
        }
    }
}

bool check(int x, int y, int col, int dir, int depth){
    x += direction[dir][0];
    y += direction[dir][1];

    if ((x < 0 || 7 < x) || (y < 0 || 7 < y) ) return false;
    if (ban[y][x] == yet) return false;

    if (ban[y][x] == col) {
        if(depth == 0) return false;
        else return true;
    }else{
        if(check(x, y, col, dir, depth+1)){
            return true;
        }else{
            return false;
        }
    }
}

bool check_xy(int x, int y, int col){
    if (ban[y][x] != 0) return false;
    bool pos = false;
    for (int i = 0; i < 8; i++) {
        pos |= check(x, y, col, i, 0);
    }
    return pos;
}

void update_xy(int x, int y, int col, int banmen[8][8]){
    if (banmen[y][x] != 0) return;
    bool pos = false;
    for (int i = 0; i < 8; i++) {
        pos |= update(x, y, col, i, 0, banmen);
    }
    if (pos) banmen[y][x] = col;
}

std::map<int, int> count(){
    std::map<int, int> m;
    m[wht] = 0;
    m[blk] = 0;
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            m[ban[i][j]]++;
        }
    }
    return m;
}

std::vector<pair<int, int> > get_putList(int col){
    vector<pair<int, int> > V;
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            if(check_xy(j, i, col)) V.push_back(make_pair(j, i));
        }
    }
    return V;
}

bool end_game(){
    return (get_putList(wht).size() == 0 && get_putList(blk).size() == 0);
}

void disp_ban(){
    std::cout << endl << ' ';

    for (int i = 0; i < 8; i++) {
        std::cout << ' ' << i;
    }

    std::cout << endl;

    for (int i = 0; i < 8; i++) {
        std::cout << i;
        for (int j = 0; j < 8; j++) {
            string s = (ban[i][j] == wht)? "◯":"◉";
            std::cout << ' ' << ((ban[i][j] != blk && ban[i][j] != wht)? "_":s);
        }
        std::cout << endl;
    }
}

class ban_hist{
public:
    vector<double> myban;
    vector<double> myans;

    ban_hist(){
        myban.resize(64);
        myans.resize(64);
    }

    void bancpy(int col){
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                int index = i*8 + j;
                if (check_xy(j, i, col)) {
                    myban[index] = 3.0;
                }else{
                    if (ban[i][j] == yet) {
                        myban[index] = 0.0;
                    }else{
                        myban[index] = double((col == wht)? ban[i][j]: ((ban[i][j] == wht)? blk: wht));
                    }
                }
            }
        }
    }

    void bancpy_separate(int pcol){
        myban.resize(128);
        for (int k = 0; k < 2; k++) {
            for (int i = 0; i < 8; i++) {
                for (int j = 0; j < 8; j++) {
                    myban[64*k+8*i+j] = (ban[i][j] == pcol)? 1.0: 0.0;
                }
            }
            pcol = (pcol == wht)? blk: wht;
        }
    }

    void anscpy(pair<int, int> p){
        int index = p.second*8 + p.first;
        fill(myans.begin(), myans.end(), 0.0);
        myans[index] = 1.0;
    }

    void print(){
        for (int i = 0; i < myban.size(); i++) {
            if (i != 0) std::cout << ' ';
            std::cout << myban[i];
        }
        std::cout << std::endl;
        for (int i = 0; i < myans.size(); i++) {
            if (i != 0) cout << ' ';
            std::cout << myans[i];
        }
        std::cout << endl;
    }

    void printmyset(){
        for (int i = 0; i < myban.size(); i++) {
            if (i%8 == 0) {
                std::cout << endl;
            }else{
                std::cout << ' ';
            }
            std::cout << myban[i];
        }
        std::cout << endl;
        for (int i = 0; i < myans.size(); i++) {
            if (i%8 == 0) {
                std::cout << endl;
            }else{
                std::cout << ' ';
            }
            std::cout << myans[i];
        }
        std::cout << std::endl;
    }
};

class nn_reader{
public:
    network net;

    nn_reader(){}

    void make_initial(){
        network n(64, 3, 128, 64, 300);
        net = n;
    }

    nn_reader(string name){
        net.load_network(name);
    }

    void reload_network(string name){
        net.load_network(name);
    }

    void save_network(string name){
        net.save_network(name);
    }

    pair<int, int> nnAnsor(int pcol){
        ban_hist b;
        b.bancpy(pcol);

        vector<double> v = net.prediction(b.myban);

        vector<pair<int, int> > plist = get_putList(pcol);
        vector<pair<pair<int ,int>, double > > pos;

        double sum = 0.0;

        for (int i = 0; i < plist.size(); i++) {
            int x = plist[i].first;
            int y = plist[i].second;
            pos.push_back(make_pair(plist[i], v[x + 8*y]));

            sum += exp(v[x + 8*y]);
        }

        for (int i = 0; i < pos.size(); i++) {
            pos[i].second = exp(pos[i].second)/sum;
        }

        double rand01 = (rand()%10000000)/10000000.0;
        sum = 0;
        int index = 0;
        for (int i = 0; i < pos.size()+5; i++) {
            sum += pos[i%pos.size()].second;
            if (sum > rand01) {
                index = i%pos.size();
                break;
            }
        }
        return pos[index].first;
    }

    pair<int, int> nnAnsorMax(int pcol){
        ban_hist b;
        b.bancpy(pcol);

        std::vector<double> v = net.prediction(b.myban);
        std::vector<std::pair<int, int> > plist = get_putList(pcol);

        std::priority_queue<pair<double, pair<int, int> > > pq;

        for (int i = 0; i < plist.size(); i++) {
            pq.push(std::make_pair(v[plist[i].first + plist[i].second*8], plist[i]));
        }

        return pq.top().second;
    }
};

class nn_reader_sp: public nn_reader{
public:
    nn_reader_sp(){}

    void make_initial(){
        network n(128, 3, 128, 2, 300);
        net = n;
    }

    nn_reader_sp(string name){
        net.load_network(name);
    }

    std::priority_queue<std::pair<double, std::pair<int, int> > > get_value_list(int pcol){
        std::vector<std::pair<int, int> > v = get_putList(pcol);
        std::priority_queue<std::pair<double, std::pair<int, int> > > anspq;

        //盤面の勝率計算をしてqueueに追加
        for (int i = 0; i < v.size(); i++) {
            int tempban[8][8];
            cpytoarray(tempban);
            update_xy(v[i].first, v[i].second, pcol, tempban);
            std::vector<double> tempvec;
            cpytovector(tempvec, tempban, pcol);
            std::vector<double> ans = net.prediction(tempvec);
            anspq.push(std::make_pair(ans[0], v[i]));
        }
        return anspq;
    }

    std::pair<int, int> nnAnsor(int pcol){
        std::priority_queue<std::pair<double, std::pair<int, int> > > anspq;
        std::vector<std::pair<double, std::pair<int, int> > > vec;

        anspq = get_value_list(pcol);

        int n = anspq.size();
        for (int i = 0; i < n; i++) {
            vec.push_back(anspq.top());
            anspq.pop();
        }
        //出力結果をsoftmax
        double sum = 0;
        for (int i = 0; i < vec.size(); i++) {
            sum += exp(vec[i].first+1);
        }
        for (int i = 0; i < vec.size(); i++) {
            vec[i].first = exp(vec[i].first+1)/sum;
        }
        //vecの中から乱択
        double rand01 = (rand()%10000000)/10000000.0;
        sum = 0;
        int index = 0;
        for (int i = 0; i < vec.size()+3; i++) {
            sum += vec[i%vec.size()].first;
            if (sum > rand01) {
                index = i%vec.size();
                break;
            }
        }
        return vec[index].second;
    }

    std::pair<int, int> nnAnsorMax(int pcol){
        std::priority_queue<std::pair<double, std::pair<int, int> > > anslist;
        anslist = get_value_list(pcol);
        return anslist.top().second;
    }
};

int digitsNumber(int num){
    int dNum = 0;
    while (num > 0) {
        num /= 10;
        dNum++;
    }
    return dNum;
}

string makeName(int game_num){
    string s;
    for (int i = 0; i < 5 - digitsNumber(game_num); i++) {
        s = s + "0";
    }
    return "othellodata/game" + s + to_string(game_num) + ".txt";
}

void vs_random(int pcolor){
    init_ban();
    disp_ban();

    int player = blk;

    while (!end_game()) {

        vector<pair<int, int> > v = get_putList(player);
        if (v.size() == 0) {
            cout << ((player == wht)? "white ": "black ") << "pass." << endl;
            player = (player == wht)? blk: wht;
            continue;
        }

        int x,y;

        if (player == pcolor) {
            std::cout << "put xy" << std::endl;
            bool flag = 1;
            while (flag) {
                int k;
                std::cin >> k;
                x = k/10;
                y = k%10;
                if (check_xy(x, y, pcolor)) {
                    flag = 0;
                }else{
                    std::cout << "you can't put there." << std::endl;
                }
            }

        }else{
            int select = rand()%v.size();
            x = v[select].first;
            y = v[select].second;
        }

        update_xy(x, y, player, ban);
        disp_ban();
        std::cout << ((player == pcolor)? "you ": "cp ") << "put " << x << y << std::endl;
        player = (player == wht)? blk: wht;
    }
}

int keta(int num,int k){
    if (num) {
        return keta(num/10, k+1);
    }else{
        return k;
    }
}

void vs_NN(int pcolor, string name){
    init_ban();
    disp_ban();
    nn_reader_sp net(name);

    int player = blk;

    while (!end_game()) {

        vector<pair<int, int> > v = get_putList(player);
        if (v.size() == 0) {
            std::cout << ((player == wht)? "white ": "black ") << "pass." << std::endl;
            player = (player == wht)? blk: wht;
            continue;
        }

        int x,y;

        if (player == pcolor) {
            std::cout << "put xy" << std::endl;
            bool flag = 1;
            while (flag) {
                int k;
                std::cin >> k;
                x = k/10;
                y = k%10;
                if (check_xy(x, y, pcolor)) {
                    flag = 0;
                }else{
                    std::cout << "you can't put there." << std::endl;
                }
            }

        }else{
            std::pair<int, int> p = net.nnAnsorMax(player);
            x = p.first;
            y = p.second;
        }

        update_xy(x, y, player, ban);
        disp_ban();
        std::cout << ((player == pcolor)? "you ": "cp ") << "put " << x << y << std::endl;
        player = (player == wht)? blk: wht;
    }

    map<int, int> counter = count();
    cout << "black: " << counter[blk] << endl;
    cout << "white: " << counter[wht] << endl;
}

bool rand_vs_nn(int randcolor, string nn_name){
    init_ban();
    nn_reader_sp net(nn_name);

    int player = blk;

    while (!end_game()) {

        std::vector<pair<int, int> > v = get_putList(player);
        if (v.size() == 0) {
            player = (player == wht)? blk: wht;
            continue;
        }

        int x,y;

        if (player == randcolor) {
            int select = rand()%v.size();
            x = v[select].first;
            y = v[select].second;
        }else{
            std::pair<int, int> p = net.nnAnsorMax(player);
            x = p.first;
            y = p.second;
        }

        update_xy(x, y, player, ban);
        player = (player == wht)? blk: wht;
    }

    std::map<int, int> counter = count();

    return counter[randcolor] < counter[((randcolor == wht)? blk: wht)];
}

void nn_vs_nn(int start_num, int end_num, string name){
    //ゲームをAIにさせた結果を保存しつつ200イテレーション、試合数は AI vs Rand 50, AI vs AI 50

    std::string nn_name = name + std::to_string(start_num-1);
    std::string nn_prename = nn_name;

    nn_reader_sp nr;

    //ファイルが存在しない場合作成
    if(start_num == 1){
      nr.make_initial();
      nr.save_network(nn_prename);
    }

    for (int sequence = start_num; sequence <= end_num; sequence++) {
        clock_t start = clock();

        nn_prename = nn_name;
        nn_name = name + std::to_string(sequence);

        //preで読み込んでnameで保存
        nr.reload_network(nn_prename);
        vector<ban_hist> win_and_d_hist;
        vector<ban_hist> lose_hist;

        for (int i = 0; i < 90; i++) {
            vector<ban_hist> temp_blackhist;
            vector<ban_hist> temp_whitehist;

            init_ban();
            int player = blk;

            while (!end_game()) {
                vector<pair<int, int> > v = get_putList(player);
                if (v.size() == 0) {
                    player = (player == wht)? blk: wht;
                    continue;
                }

                ban_hist hist;
                pair<int, int> p;

                if (sequence != 1) {
                    p = nr.nnAnsor(player);
                }else{
                    p = v[rand()%v.size()];
                }

                hist.bancpy_separate(player);
                update_xy(p.first, p.second, player, ban);

                if (player == blk) {
                    temp_blackhist.push_back(hist);
                }else{
                    temp_whitehist.push_back(hist);
                }

                player = (player == wht)? blk: wht;
            }
            std::map<int, int> counter = count();

            //データセット分類
            if (counter[blk] == counter[wht]) {
                win_and_d_hist.insert(win_and_d_hist.end(), temp_blackhist.begin(), temp_blackhist.end());
                win_and_d_hist.insert(win_and_d_hist.end(), temp_whitehist.begin(), temp_whitehist.end());
            }else if (counter[blk] > counter[wht]) {
                win_and_d_hist.insert(win_and_d_hist.end(), temp_blackhist.begin(), temp_blackhist.end());
                lose_hist.insert(lose_hist.end(), temp_whitehist.begin(), temp_whitehist.end());
            }else{
                win_and_d_hist.insert(win_and_d_hist.end(), temp_whitehist.begin(), temp_whitehist.end());
                lose_hist.insert(lose_hist.end(), temp_blackhist.begin(), temp_blackhist.end());
            }

        }
        //ここでRandom に対する負け試合を50 得たい。

        int lose_counter = 0;
        int rand_color = blk;
        int rand_rate_count = 0;

        while (lose_counter < 10) {
            rand_rate_count++;
            vector<ban_hist> temp_blackhist;
            vector<ban_hist> temp_whitehist;

            init_ban();
            int player = blk;
            rand_color = (rand_color == wht)? blk: wht;

            while (!end_game()) {
                vector<pair<int, int> > v = get_putList(player);
                if (v.size() == 0) {
                    player = (player == wht)? blk: wht;
                    continue;
                }

                ban_hist hist;
                pair<int, int> p;

                if (player == rand_color) {
                    p = v[rand()%v.size()];
                }else{
                    p = nr.nnAnsorMax(player);
                }

                hist.bancpy_separate(player);
                update_xy(p.first, p.second, player, ban);

                if (player == blk) {
                    temp_blackhist.push_back(hist);
                }else{
                    temp_whitehist.push_back(hist);
                }

                player = (player == wht)? blk: wht;
            }
            std::map<int, int> counter = count();

            //random に負けた場合のみデータセットに追加
            if (counter[rand_color] > counter[((rand_color == blk)? wht: blk)]) {
                if (rand_color == blk) {
                    win_and_d_hist.insert(win_and_d_hist.end(), temp_blackhist.begin(), temp_blackhist.end());
                    lose_hist.insert(lose_hist.end(), temp_whitehist.begin(), temp_whitehist.end());
                }else{
                    win_and_d_hist.insert(win_and_d_hist.end(), temp_whitehist.begin(), temp_whitehist.end());
                    lose_hist.insert(lose_hist.end(), temp_blackhist.begin(), temp_blackhist.end());
                }
                lose_counter++;
            }
        }

        cout << "win_rate = " << (1.0*(rand_rate_count - 10))/rand_rate_count << endl;

        //ここまでで学習データ作成完了。

        vector<vector<double> > matban;
        vector<vector<double> > matans;

        for (int i = 0; i < win_and_d_hist.size(); i++) {
            matban.push_back(win_and_d_hist[i].myban);
            std::vector<double> ans(2, 0.0);
            ans[0] = 1.0;
            matans.push_back(ans);
        }

        for (int i = 0; i < lose_hist.size(); i++) {
            matban.push_back(lose_hist[i].myban);
            std::vector<double> ans(2, 0.0);
            ans[1] = 1.0;
            matans.push_back(ans);
        }

        matplotlib g;
        g.open();
        g.screen(0, 0, 200, 1);

        double prime = 0.0;
        for (int i = 0; i < 200; i++) {
            nr.net.for_and_backward(matban, matans);
            nr.net.leaning_adam(0.001);
            double err = nr.net.calculate_error(matban, matans);
            std::cout << "sequence:" << sequence << " " << i+1 << " err = " << err << std::endl;
            g.line(i-1,prime,i,err);
            prime = err;
        }

        g.close();
        nr.net.save_network(nn_name);
//        int game_counter = 0;
//        int col = wht;
//        for (int game = 0; game < 300; game++) {
//            game_counter += rand_vs_nn(col, nn_name);
//            col = (col == wht)? blk: wht;
//        }
//        cout << "win_rate = " << game_counter/300.0 << endl;
        clock_t end = clock();
        std::cout << "sequence " << sequence << " end in " << (double)(end - start) / CLOCKS_PER_SEC << "sec." << std::endl;
    }
}

void init(){
    srand((unsigned int)time(0));

    color_string[wht] = "white";
    color_string[blk] = "black";
}

double evale_nn(string name){
    clock_t start = clock();
    int col = blk;
    int counter = 0;
    for (int i = 0; i < 300; i++) {
        counter += rand_vs_nn(col, name);
        col = (col == blk)? wht: blk;
    }
    clock_t end = clock();
    std::cout << name << " takes " << (double)(end - start) / CLOCKS_PER_SEC << "sec." << std::endl;
    return counter/300.0;
}

int main(){
    init();

    nn_vs_nn(1, 2, "fromLinux");

    return 0;
}
