#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>

#define BLOCK_SIZE 16
#define SIZE_RATE 10

struct matrix{
  int height;
  int width;
  float *elements;
};

void matrixSizeInit(matrix M, int h, int w){
  M.height = h;
  M.width = w;

  int size = h*w;

  delete [] M.elements;
  M.elements = new float[size];
  for(int i = 0; i < size; i++) M.elements[i] = 0.0;
}

void sizeInitFromMatrix(matrix M, matrix from){
  M.height = from.height;
  M.width = from.width;

  int size = from.height*from.width;

  delete [] M.elements;
  M.elements = new float[size];
  for(int i = 0; i < size; i++) M.elements[i] = 0.0;
}

void printMatrix(matrix M){
  for(int i = 0; i < M.height; i++){
    for(int j = 0; j < M.width; j++){
      if(j != 0) std::cout << " ";
      std::cout << M.elements[i*M.width+j];
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

__global__ void matrixAdd(matrix M, matrix add){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] += add.elements[row*add.width+col];
  }
}

__global__ void matrixMul(matrix A, matrix B, matrix C){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < C.height && col < C.width){
    float x = 0.0f;
    for (int i = 0; i < A.width; i++) {
      x += A.elements[row*A.width+i]*B.elements[i*B.width+col];
    }

    C.elements[row*C.width+col] = x;
  }
}

static void matrixMul_gpu(matrix& d_m_in, matrix& d_m_ac){
  //デバイスに演算結果の領域を確保
  matrix d_ans;
  d_ans.width = d_m_ac.width; d_ans.height = d_m_in.height;

  int size;
  //デバイスにメモリ確保
  size = d_ans.width*d_ans.height*sizeof(float);
  hipMalloc((void**)&d_ans.elements, size);

  //Cのサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_ans.width-1+blk.x)/blk.x, (d_ans.height-1+blk.y)/blk.y);

  matrixMul<<<gld, blk>>>(d_m_in, d_m_ac, d_ans);

  matrix A;
  A.height = 2;
  A.width = 2;
  A.elements = new float[A.height*A.width];
  hipMemcpy(A.elements, d_m_ac.elements, size, hipMemcpyDeviceToHost);
  std::cout << "matrix:actcpy =" << std::endl;
  printMatrix(A);

  //不要になった入力のメモリの開放
  hipFree(d_m_in.elements);

  //演算結果を引き継ぐ
  d_m_in = d_ans;
}

__global__ void matrixAddBias(matrix M, matrix bias){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] += bias.elements[col];
  }
}

__global__ void matrixRelu(matrix M){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] = (M.elements[row*M.width+col] < 0)? 0: M.elements[row*M.width+col];
  }
}

void randomInit(matrix m, int maxVal){
  for(int i = 0; i < m.height*m.width; i++) m.elements[i] = maxVal*(float(rand())/RAND_MAX) - maxVal/2.0;
}

__gloval__ void name(matrix a) {

}

void checkFunction(){
  matrix A, B;
  A.height = A.width = 2;
  B.height = B.width = 2;

  A.elements = new float[A.width*A.height];
  B.elements = new float[B.width*B.height];

  randomInit(A, 10);
  randomInit(B, 10);

  //演算前確認
  std::cout << "matrix:A =" << std::endl;
  printMatrix(A);
  std::cout << "matrix:B =" << std::endl;
  printMatrix(B);

  matrix dA, dB;
  dA.width = A.width; dA.height = A.height;
  dB.width = B.width; dB.height = B.height;

  int a;
  &a

  int *a;
  *a = 1;
  a = 1が入ってるアドレス
  int b = 2;
  a = &b;
  &a//=2

  int size = dA.width*dA.height*sizeof(float);
  hipMalloc((void**)&dA.elements, size);
  hipMemcpy(dA.elements, A.elements, size, hipMemcpyHostToDevice);

  hipMemcpy(B.elements, dA.elements, size, hipMemcpyDeviceToHost);

  std::cout << "matrix:B =" << std::endl;
  printMatrix(B);

  // size = dB.width*dB.height*sizeof(float);
  // hipMalloc((void**)&dB.elements, size);
  // hipMemcpy(dB.elements, B.elements, size, hipMemcpyHostToDevice);
  //
  // func(dA, dB);
  //
  // hipMemcpy(A.elements, dA.elements, size, hipMemcpyDeviceToHost);
  // std::cout << "matrix:ans =" << std::endl;
  // printMatrix(A);
  //
  // // ホストメモリ解放
  // delete [] A.elements;
  // delete [] B.elements;
}

int main(){
  checkFunction();
  return 0;
}
