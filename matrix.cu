#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#define BLOCK_SIZE 16
#define SIZE_RATE 10

struct matrix{
  int height;
  int width;
  float *elements;
};

void matrixSizeInit(matrix M, int h, int w){
  M.height = h;
  M.width = w;

  int size = h*w;

  delete [] M.elements;
  M.elements = new float[size];
  for(int i = 0; i < size; i++) M.elements[i] = 0.0;
}

void sizeInitFromMatrix(matrix M, matrix from){
  M.height = from.height;
  M.width = from.width;

  int size = from.height*from.width;

  delete [] M.elements;
  M.elements = new float[size];
  for(int i = 0; i < size; i++) M.elements[i] = 0.0;
}

void printMatrix(matrix M){
  for(int i = 0; i < M.height; i++){
    for(int j = 0; j < M.width; j++){
      if(j != 0) std::cout << " ";
      std::cout << M.elements[i*M.width+j];
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

__global__ void matrixCpy_cuda(matrix M, matrix org){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] = org.elements[row*org.width+col];
  }
}

static void matrixCpy(matrix& d_m_in, matrix& d_m_ac){
  //入力のサイズをコピー元と合わせる。
  hipFree(d_m_in.elements);
  d_m_in.height = d_m_ac.height; d_m_in.width = d_m_ac.width;
  hipMalloc((void**)&d_m_in.elements, d_m_in.height*d_m_in.width*sizeof(float));
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixCpy_cuda<<<gld, blk>>>(d_m_in, d_m_ac);
}

__global__ void matrixAdd_cuda(matrix M, matrix add){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] += add.elements[row*add.width+col];
  }
}

static void matrixAdd(matrix& d_m_in, matrix& d_m_ac){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixAdd_cuda<<<gld, blk>>>(d_m_in, d_m_ac);
}

__global__ void matrixMinus_cuda(matrix M, matrix minus){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] -= minus.elements[row*minus.width+col];
  }
}

static void matrixMinus(matrix& d_m_in, matrix& d_m_ac){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixMinus_cuda<<<gld, blk>>>(d_m_in, d_m_ac);
}

__global__ void matrixConstMul_cuda(matrix M, int rate){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] *= rate;
  }
}

static void matrixConstMul(matrix& d_m_in, int rate){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixConstMul_cuda<<<gld, blk>>>(d_m_in, rate);
}

__global__ void matrixMul_cuda(matrix A, matrix B, matrix C){
  //行列Cにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < C.height && col < C.width){
    float x = 0.0f;
    for (int i = 0; i < A.width; i++) {
      x += A.elements[row*A.width+i]*B.elements[i*B.width+col];
    }

    C.elements[row*C.width+col] = x;
  }
}

static void matrixMul(matrix& d_m_in, matrix& d_m_ac){
  //デバイスに演算結果の領域を確保
  matrix d_ans;
  d_ans.width = d_m_ac.width; d_ans.height = d_m_in.height;

  int size;
  //デバイスにメモリ確保
  size = d_ans.width*d_ans.height*sizeof(float);
  hipMalloc((void**)&d_ans.elements, size);

  //Cのサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_ans.width-1+blk.x)/blk.x, (d_ans.height-1+blk.y)/blk.y);

  matrixMul_cuda<<<gld, blk>>>(d_m_in, d_m_ac, d_ans);

  //不要になった入力のメモリの開放
  hipFree(d_m_in.elements);

  //演算結果を引き継ぐ
  d_m_in = d_ans;
}

__global__ void matrixAddBias_cuda(matrix M, matrix bias){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] += bias.elements[col];
  }
}

static void matrixAddBias(matrix& d_m_in, matrix& d_m_ac){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixAddBias_cuda<<<gld, blk>>>(d_m_in, d_m_ac);
}

__global__ void matrixRelu_cuda(matrix M){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < M.height && col < M.width){
    M.elements[row*M.width+col] = (M.elements[row*M.width+col] < 0)? 0: M.elements[row*M.width+col];
  }
}

static void matrixRelu(matrix& d_m_in){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_in.width-1+blk.x)/blk.x, (d_m_in.height-1+blk.y)/blk.y);

  matrixRelu_cuda<<<gld, blk>>>(d_m_in);
}

__global__ void matrixTranspose_cuda(matrix M, matrix trans){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < trans.height && col < trans.width){
    trans.elements[row*trans.width+col] = M.elements[col*M.width+row];
  }
}

static void matrixTranspose(matrix& d_m_in){
  //デバイスに演算結果の領域を確保
  matrix d_ans;
  d_ans.height = d_m_in.width; d_ans.width = d_m_in.height;
  int size = d_ans.width*d_ans.height*sizeof(float);
  hipMalloc((void**)&d_ans.elements, size);

  //Cのサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_ans.width-1+blk.x)/blk.x, (d_ans.height-1+blk.y)/blk.y);

  matrixTranspose_cuda<<<gld, blk>>>(d_m_in, d_ans);

  //不要になった入力のメモリの開放
  hipFree(d_m_in.elements);

  //演算結果を引き継ぐ
  d_m_in = d_ans;
}

__global__ void matrixWithFunc1_cuda(matrix m1, void (*func)(float&)){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < m1.height && col < m1.width){
    func(&m1.elements[row*m1.width+col]);
  }
}

static void matrixFunc1(matrix& d_m_1, void (*func)(float&)){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_1.width-1+blk.x)/blk.x, (d_m_1.height-1+blk.y)/blk.y);

  matrixFunc1_cuda<<<gld, blk>>>(d_m_1, func);
}

__global__ void matrixWithFunc2_cuda(matrix m1, matrix m2, void (*func)(float&, float&)){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < m1.height && col < m1.width){
    func(&m1.elements[row*m1.width+col], &m2.elements[row*m2.width+col]);
  }
}

static void matrixFunc2(matrix& d_m_1, matrix& d_m_2, void (*func)(float&, float&)){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_1.width-1+blk.x)/blk.x, (d_m_1.height-1+blk.y)/blk.y);

  matrixFunc2_cuda<<<gld, blk>>>(d_m_1, d_m_2, func);
}

__global__ void matrixWithFunc3_cuda(matrix m1, matrix m2, matrix m3, void (*func)(float&, float&, float&)){
  //行列Mにおけるどこを計算するスレッドか確定する。
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;

  //計算が必要なスレッドか確認
  if(row < m1.height && col < m1.width){
    func(&m1.elements[row*m1.width+col], &m2.elements[row*m2.width+col], &m3.elements[row*m3.width+col]);
  }
}

static void matrixFunc3(matrix& d_m_1, matrix& d_m_2, matrix& d_m_3, void (*func)(float&, float&, float&)){
  //入力のサイズに合わせてブロックとグリッドの設定
  dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gld((d_m_1.width-1+blk.x)/blk.x, (d_m_1.height-1+blk.y)/blk.y);

  matrixFunc3_cuda<<<gld, blk>>>(d_m_1, d_m_2, d_m_3, func);
}

void randomInit(matrix m, int maxVal){
  for(int i = 0; i < m.height*m.width; i++) m.elements[i] = int(maxVal*(float(rand())/RAND_MAX)) - maxVal/2.0;
}

void checkFunction(void (*func)(matrix&, matrix&), int ah, int aw, int bh, int bw){
  matrix A, B;
  A.height = ah; A.width = aw;
  B.height = bh; B.width = bw;

  A.elements = new float[A.width*A.height];
  B.elements = new float[B.width*B.height];

  randomInit(A, 10);
  randomInit(B, 10);

  //演算前確認
  std::cout << "matrix:in(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);
  std::cout << "matrix:act(" << B.height << ", " << B.width << ") =" << std::endl;
  printMatrix(B);

  matrix dA, dB;
  dA.width = A.width; dA.height = A.height;
  dB.width = B.width; dB.height = B.height;

  int size = dA.width*dA.height*sizeof(float);
  hipMalloc((void**)&dA.elements, size);
  hipMemcpy(dA.elements, A.elements, size, hipMemcpyHostToDevice);

  size = dB.width*dB.height*sizeof(float);
  hipMalloc((void**)&dB.elements, size);
  hipMemcpy(dB.elements, B.elements, size, hipMemcpyHostToDevice);

  func(dA, dB);

  //Aのサイズを変更されたdAのサイズに合わせる。
  delete [] A.elements;
  A.height = dA.height;
  A.width = dA.width;
  A.elements = new float[A.height*A.width];
  size = dA.width*dA.height*sizeof(float);
  hipMemcpy(A.elements, dA.elements, size, hipMemcpyDeviceToHost);

  std::cout << "matrix:ans(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);

  // ホストメモリ解放
  delete [] A.elements;
  delete [] B.elements;

  for(int i = 0; i < 25; i++) std::cout << "-";
  std::cout << std::endl;
}

void checkFunction2(void (*func)(matrix&), int ah, int aw){
  //行列作成
  matrix A;
  A.height = ah; A.width = aw;
  A.elements = new float[A.width*A.height];
  randomInit(A, 10);

  //演算前確認
  std::cout << "matrix:in(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);

  matrix dA;
  dA.width = A.width; dA.height = A.height;

  int size = dA.width*dA.height*sizeof(float);
  hipMalloc((void**)&dA.elements, size);
  hipMemcpy(dA.elements, A.elements, size, hipMemcpyHostToDevice);

  func(dA);

  //Aのサイズを変更されたdAのサイズに合わせる。
  delete [] A.elements;
  A.height = dA.height;
  A.width = dA.width;
  A.elements = new float[A.height*A.width];
  size = dA.width*dA.height*sizeof(float);
  hipMemcpy(A.elements, dA.elements, size, hipMemcpyDeviceToHost);

  std::cout << "matrix:ans(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);

  // ホストメモリ解放
  delete [] A.elements;

  for(int i = 0; i < 25; i++) std::cout << "-";
  std::cout << std::endl;
}

void checkFunction3(void (*func)(matrix&, int), int ah, int aw, int rate){
  //行列作成
  matrix A;
  A.height = ah; A.width = aw;
  A.elements = new float[A.width*A.height];
  randomInit(A, 10);

  //演算前確認
  std::cout << "matrix:in(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);

  matrix dA;
  dA.width = A.width; dA.height = A.height;

  int size = dA.width*dA.height*sizeof(float);
  hipMalloc((void**)&dA.elements, size);
  hipMemcpy(dA.elements, A.elements, size, hipMemcpyHostToDevice);

  func(dA, rate);

  //Aのサイズを変更されたdAのサイズに合わせる。
  delete [] A.elements;
  A.height = dA.height;
  A.width = dA.width;
  A.elements = new float[A.height*A.width];
  size = dA.width*dA.height*sizeof(float);
  hipMemcpy(A.elements, dA.elements, size, hipMemcpyDeviceToHost);

  std::cout << "matrix:ans(" << A.height << ", " << A.width << ") =" << std::endl;
  printMatrix(A);

  // ホストメモリ解放
  delete [] A.elements;

  for(int i = 0; i < 25; i++) std::cout << "-";
  std::cout << std::endl;
}

void checkAll(){
  std::cout << "cpy" << std::endl;
  checkFunction(matrixCpy, 3,3,2,2);
  std::cout << "add" << std::endl;
  checkFunction(matrixAdd, 2,2,2,2);
  std::cout << "minus" << std::endl;
  checkFunction(matrixMinus, 2,2,2,2);
  std::cout << "mul" << std::endl;
  checkFunction(matrixMul, 2,3,3,2);
  std::cout << "bias" << std::endl;
  checkFunction(matrixAddBias, 2,3,1,3);
  std::cout << "relu" << std::endl;
  checkFunction2(matrixRelu, 2,3);
  std::cout << "trans" << std::endl;
  checkFunction2(matrixTranspose, 2,3);
  std::cout << "const Mul " << 2 << std::endl;
  checkFunction3(matrixConstMul, 2,2,2);//最後の引数は倍率ß
}

int main(){
  checkAll();
  return 0;
}
